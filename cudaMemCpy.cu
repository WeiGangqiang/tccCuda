#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>

int main() {
    const size_t dataSize = 1024 * 972 * 8; // 例如，256MB
    char* hostData;
    char* deviceData;

    hipHostMalloc(&hostData, dataSize, hipHostMallocDefault); // 使用页锁定内存
    hipMalloc(&deviceData, dataSize);

    auto start = std::chrono::high_resolution_clock::now();

    hipMemcpy(deviceData, hostData, dataSize, hipMemcpyHostToDevice);

    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double, std::milli> duration = end - start;

    std::cout << "Transfer time: " << duration.count() << " ms\n";

    hipHostFree(hostData);
    hipFree(deviceData);

    return 0;
}